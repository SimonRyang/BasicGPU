
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void maxPoly(const double x0, const double* coef,
			const double tol, const int nParam, double* argMax){

  // Thread ID
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  // The Kernel should only execute if i < nParam
  if(i >= nParam){
    return;
  } else {

    // Iterate to convergence
    double x = x0;
    double diff = tol+1;
    double firstDeriv, secondDeriv, xNew;
    while(diff > tol){
      
      // Compute the first derivative
      firstDeriv = 2*coef[i]*x + 2.3;
      
      // Compute the second derivative
      secondDeriv = 2*coef[i];
      
      // Newton step
      xNew = x - firstDeriv/secondDeriv;
      
      // Compute difference for convergence check and update
      diff = fabs(xNew - x);
      x = xNew;
      
    }

    // Function outpout
    argMax[i] = x;
  }

}
