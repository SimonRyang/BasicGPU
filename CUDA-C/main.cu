#include "maxPoly.cu"

int main()
{

  // Grid for order 2 coefficient
  int nParam = 1000000000;
  double paramMin = 0.1;
  double paramMax = 0.9;
  double* paramGrid = new double[nParam];
  for(int i = 0 ; i < nParam ; ++i) paramGrid[i] = paramMin + i*(paramMax-paramMin)/(nParam-1);

  // Copy parameter grid from CPU to GPU memory
  double* paramGridDevice;
  hipMalloc((void**)&paramGridDevice, nParam*sizeof(double));
  hipMemcpy(paramGridDevice, paramGrid, nParam*sizeof(double), hipMemcpyHostToDevice);

  // Storage for argmax values
  double* argMaxValsDevice;
  hipMalloc((void**)&argMaxValsDevice, nParam*sizeof(double));

  // Maximize for each coefficient
  maxPoly<<<1,256>>>(2.2, paramGridDevice, 0.00001, argMaxValsDevice);

  // Copy argmax values from GPU to CPU memory
  double* argMaxVals = new double[nParam];
  hipMemcpy(argMaxVals, argMaxValsDevice, nParam*sizeof(double), hipMemcpyDeviceToHost);

  return 0;

}
