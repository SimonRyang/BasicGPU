#include "maxPoly.cu"
#include <iostream>
using namespace std;

int main()
{

  // Grid for order 2 coefficient
  int nParam = 1024;
  double paramMin = -0.9;
  double paramMax = -0.1;
  double* paramGrid = new double[nParam];
  for(int i = 0 ; i < nParam ; ++i) paramGrid[i] = paramMin + i*(paramMax-paramMin)/(nParam-1);

  // Copy parameter grid from CPU to GPU memory
  double* paramGridDevice;
  hipError_t test = hipMalloc((void**)&paramGridDevice, nParam*sizeof(double));
  cout << test << endl;
  hipMemcpy(paramGridDevice, paramGrid, nParam*sizeof(double), hipMemcpyHostToDevice);

  // Storage for argmax values
  double* argMaxValsDevice;
  hipMalloc((void**)&argMaxValsDevice, nParam*sizeof(double));

  // Maximize for each coefficient
  int blockLength = 256;
  int gridLength = nParam/blockLength;
  maxPoly<<<gridLength,blockLength>>>(2.2, paramGridDevice, 0.00001, argMaxValsDevice);

  // Copy argmax values from GPU to CPU memory
  double* argMaxVals = new double[nParam];
  hipMemcpy(argMaxVals, argMaxValsDevice, nParam*sizeof(double), hipMemcpyDeviceToHost);

  return 0;

}
