#include "maxPoly.cu"

int main()
{

  // Grid for order 2 coefficient
  int nParam = 1024;
  double paramMin = -0.9;
  double paramMax = -0.1;
  double* paramGrid = new double[nParam];
  for(int i = 0 ; i < nParam ; ++i) paramGrid[i] = paramMin + i*(paramMax-paramMin)/(nParam-1);

  // Copy parameter grid from CPU to GPU memory
  double* paramGridDevice;
  hipMalloc((void**)&paramGridDevice, nParam*sizeof(double));
  hipMemcpy(paramGridDevice, paramGrid, nParam*sizeof(double), hipMemcpyHostToDevice);

  // Storage for argmax values
  double* argMaxValsDevice;
  hipMalloc((void**)&argMaxValsDevice, nParam*sizeof(double));

  // Maximize for each coefficient
  int blockLength = 256;
  int gridLength = nParam/blockLength;
  maxPoly<<<gridLength,blockLength>>>(2.2, paramGridDevice, 0.00001, argMaxValsDevice);

  // Copy argmax values from GPU to CPU memory
  double* argMaxVals = new double[nParam];
  hipMemcpy(argMaxVals, argMaxValsDevice, nParam*sizeof(double), hipMemcpyDeviceToHost);

  return 0;

}
