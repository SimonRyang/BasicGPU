#include "hip/hip_runtime.h"
#include <iostream>
#include "maxPoly.cu"

using namespace std;

int main()
{

  // Grid for order 2 coefficient
  int nParam = 1024;
  double paramMin = -0.9;
  double paramMax = -0.1;
  double* paramGrid = new double[nParam];
  for(int i = 0 ; i < nParam ; ++i) paramGrid[i] = paramMin + i*(paramMax-paramMin)/(nParam-1);

  // Copy parameter grid from CPU to GPU memory
  double* paramGridDevice;
  hipMalloc((void**)&paramGridDevice, nParam*sizeof(double));
  hipMemcpy(paramGridDevice, paramGrid, nParam*sizeof(double), hipMemcpyHostToDevice);

  // Storage for argmax values
  double* argMaxValsDevice;
  hipMalloc((void**)&argMaxValsDevice, nParam*sizeof(double));

  // Maximize for each coefficient
  int threadsPerBlock = 256;
  int blocksPerGrid = (int)ceil((double)nParam/threadsPerBlock);
  maxPoly<<<blocksPerGrid, threadsPerBlock>>>(2.2, paramGridDevice,
					      0.00001, nParam, argMaxValsDevice);

  // Copy argmax values from GPU to CPU memory
  double* argMaxVals = new double[nParam];
  hipMemcpy(argMaxVals, argMaxValsDevice, nParam*sizeof(double), hipMemcpyDeviceToHost);

  for(int i = 0 ; i < nParam ; ++i){
    cout << argMaxVals[i] << endl;
  }

  return 0;

}
