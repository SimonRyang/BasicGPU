#include "hip/hip_runtime.h"
#include "maxPoly.cu"

int main()
{

  // Grid for order 2 coefficient
  int nParam = 1024;
  double paramMin = -0.9;
  double paramMax = -0.1;
  double* paramGrid = new double[nParam];
  for(int i = 0 ; i < nParam ; ++i) paramGrid[i] = paramMin + i*(paramMax-paramMin)/(nParam-1);

  // Copy parameter grid from CPU to GPU memory
  double* paramGridDevice;
  hipMalloc((void**)&paramGridDevice, nParam*sizeof(double));
  hipMemcpy(paramGridDevice, paramGrid, nParam*sizeof(double), hipMemcpyHostToDevice);

  // Storage for argmax values
  double* argMaxValsDevice;
  hipMalloc((void**)&argMaxValsDevice, nParam*sizeof(double));

  // Maximize for each coefficient
  int threadsPerBlock = 256;
  int blocksPerGrid = (int)ceil((double)nParam/threadsPerBlock);
  maxPoly<<<blocksPerGrid, threadsPerBlock>>>(2.2, paramGridDevice,
					      0.00001, nParam, argMaxValsDevice);

  // Copy argmax values from GPU to CPU memory
  double* argMaxVals = new double[nParam];
  hipMemcpy(argMaxVals, argMaxValsDevice, nParam*sizeof(double), hipMemcpyDeviceToHost);

  return 0;

}
