
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void maxPoly(double x0, double* coef,
			double tol, int nParam, double* argMax){

  // Thread ID
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  // The Kernel should only execute if i < nParam
  if(i >= nParam){
    return;
  } else {

    // Iterate to convergence
    double x = x0;
    double diff = tol+1;
    double firstDeriv, secondDeriv, xNew;
    while(diff > tol){
      
      // Compute the first derivative
      firstDeriv = 2*coef[i]*x + 2.3;
      
      // Compute the second derivative
      secondDeriv = 2*coef[i];
      
      // Newton step
      xNew = x - firstDeriv/secondDeriv;
      
      // Compute difference for convergence check and update
      diff = fabs(xNew - x);
      x = xNew;
      
    }

    // Function outpout
    argMax[i] = x;
  }

}
