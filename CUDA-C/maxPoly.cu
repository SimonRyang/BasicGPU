
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void maxPoly(double x0, double* coef, double tol, double* argMax){

  // Thread ID
  int i = threadIdx.x;

  // Iterate to convergence
  double x = x0;
  double diff = tol+1;
  double firstDeriv, secondDeriv, xNew;
  while(diff > tol){
    
    // Compute the first derivative
    firstDeriv = 2*coef[i]*x + 2.3;
         
    // Compute the second derivative
    secondDeriv = 2*coef[i];
         
    // Newton step
    xNew = x - firstDeriv/secondDeriv;
         
    // Compute difference for convergence check and update
    diff = fabs(xNew - x);
    x = xNew;
                  
  }
     
  // Function outpout
  argMax[i] = x;

}
